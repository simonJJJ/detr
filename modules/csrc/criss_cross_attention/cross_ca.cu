#include "hip/hip_runtime.h"
#include "common_cuda_helper.hpp"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


inline int GET_BLOCKS_CUSTOM(const int N, const int num_threads)
{
  return (N + num_threads - 1) / num_threads;
}


template <typename scalar_t>
__device__ scalar_t cc_bilinear(const scalar_t* input, const int height,
                                const int width, scalar_t y, scalar_t x,
                                const int index) {
  int y_low = floor(y);
  int x_low = floor(x);
  int y_high = y_low + 1;
  int x_high = y_high + 1;

  const scalar_t ly = y - y_low;
  const scalar_t lx = x - x_low;
  const scalar_t hy = 1 - ly, hx = 1 - lx;
  // do bilinear interpolation
  scalar_t v1 = 0;
  //if (y_low >= 0 && x_low >= 0)
  //{
  v1 = input[y_low * width + x_low];
  //}
  scalar_t v2 = 0;
  //if (y_low >= 0 && x_high <= width - 1)
  //{
  v2 = input[y_low * width + x_high];
  //}
  scalar_t v3 = 0;
  //if (y_high <= height - 1 && x_low >= 0)
  //{
  v3 = input[y_high * width + x_low];
  //}
  scalar_t v4 = 0;
  //if (y_high <= height - 1 && x_high <= width - 1)
  //{
  v4 = input[y_high * width + x_high];
  //}
  scalar_t w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}


template <typename scalar_t>
__device__ void cc_bilinear_gradient(const scalar_t* &bottom_data, 
                                     const int &height, const int &width,
                                     const scalar_t &h, const scalar_t &w,
                                     const scalar_t &top_grad,
                                     const scalar_t &attn_weight,
                                     scalar_t* &grad_value,
                                     scalar_t* grad_weight,
                                     scalar_t* grad_point)
{
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  const scalar_t top_grad_value = top_grad * attn_weight;
  scalar_t grad_h_weight = 0, grad_w_weight = 0;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
  {
    v1 = bottom_data[h_low * width + w_low];
    grad_h_weight -= hw * v1;
    grad_w_weight -= hh * v1;
    atomicAdd(grad_value + h_low * width + w_low, w1*top_grad_value);
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
  {
    v2 = bottom_data[h_low * width + w_high];
    grad_h_weight -= lw * v2;
    grad_w_weight += hh * v2;
    atomicAdd(grad_value + h_low * width + w_high, w2*top_grad_value);
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
  {
    v3 = bottom_data[h_high * width + w_low];
    grad_h_weight += hw * v3;
    grad_w_weight -= lh * v3;
    atomicAdd(grad_value + h_high * width + w_low, w3*top_grad_value); 
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
  {
    v4 = bottom_data[h_high * width + w_high];
    grad_h_weight += lw * v4;
    grad_w_weight += lh * v4;
    atomicAdd(grad_value + h_high * width + w_high, w4*top_grad_value);
  }

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  *grad_weight = top_grad * val;
  *grad_point = width * grad_w_weight * top_grad_value;
  *(grad_point + 1) = height * grad_h_weight * top_grad_value;
}


template <typename scalar_t>
__global__ void cross_ca_weight_forward_kernel(const int n,
                                               const scalar_t *query,
                                               const scalar_t *key,
                                               const scalar_t *point,
                                               scalar_t *output,
                                               const int batch_size,
                                               const int num_heads,
                                               const int channels,
                                               const int height,
                                               const int width,
                                               const int num_query) {
  const int len_weight = (height + width - 1);
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int weight_col = index % len_weight;
    const int head_col = (index / len_weight) % num_heads;
    const int query_col = (index / len_weight / num_heads) % num_query;
    const int batch_col = (index / len_weight / num_heads / num_query) % batch_size;

    scalar_t *output_ptr = output + index;
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    int key_base_ptr = batch_col * num_heads * channels * sp_offset + head_col * channels * sp_offset;
    int query_base_ptr = batch_col * num_query * num_heads * channels + query_col * num_heads * channels + head_col * channels;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);
    scalar_t x = 0;
    scalar_t y = 0;

    if (weight_col < width) {
      x = start_x + weight_col - 0.5;
      y = point_y - 0.5;
    } else {
      int h_index = weight_col - width;
      y = start_y + h_index;
      int j = y < point_y ? h_index : h_index + 1;
      x = point_x - 0.5;
      y = start_y + j - 0.5;
    }
    scalar_t col = 0;
    for (int plane = 0; plane < channels; ++plane) {
      const scalar_t* offset_key = key + key_base_ptr + plane * sp_offset;
      scalar_t key_inter = bilinear_interpolate(offset_key, height, width, y, x, index);
      scalar_t query_inter = query[query_base_ptr + plane];
      col += key_inter * query_inter;
    }
    *output_ptr = col;
  }
}


template <typename scalar_t>
__global__ void cross_ca_weight_backward_kernel_v1(const int n,
                                                   const scalar_t *grad_w,
                                                   const scalar_t *query,
                                                   const scalar_t *key,
                                                   const scalar_t *point,
                                                   scalar_t *grad_query,
                                                   scalar_t *grad_key,
                                                   scalar_t *grad_point,
                                                   const int batch_size,
                                                   const int num_heads,
                                                   const int channels,
                                                   const int height,
                                                   const int width,
                                                   const int num_query) {
  const int len_weight = (height + width - 1);
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    extern __shared__ int _s[];
    scalar_t* cache_grad_points = (scalar_t*)_s;
    scalar_t* cache_grad_query = cache_grad_points + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    const int weight_col = index % len_weight;
    const int head_col = (index / len_weight) % num_heads;
    const int query_col = (index / len_weight / num_heads) % num_query;
    const int batch_col = (index / len_weight / num_heads / num_query) % batch_size;

    const scalar_t top_grad = grad_w[index];
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    int key_base_ptr = batch_col * num_heads * channels * sp_offset + head_col * channels * sp_offset;
    int query_base_ptr = batch_col * num_query * num_heads * channels + query_col * num_heads * channels + head_col * channels;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);
    scalar_t x = 0;
    scalar_t y = 0;

    if (weight_col < width) {
      x = start_x + weight_col - 0.5;
      y = point_y - 0.5;
    } else {
      int h_index = weight_col - width;
      y = start_y + h_index;
      int j = y < point_y ? h_index : h_index + 1;
      x = point_x - 0.5;
      y = start_y + j - 0.5;
    }
    scalar_t grad_point_w = 0;
    scalar_t grad_point_h = 0;
    for (int plane = 0; plane < channels; ++plane) {
      const int offset_key = key_base_ptr + plane * sp_offset;
      const int offset_query = query_base_ptr + plane;
      const scalar_t *key_ptr = key + offset_key;
      scalar_t *grad_key_ptr = grad_key + offset_key;
      const scalar_t query_val = query[offset_query];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_query + threadIdx.x) = 0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(key_ptr, height, width, y, x, top_grad, query_val,
                             grad_key_ptr, cache_grad_query + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }
      __syncthreads();

      if (tid == 0) {
        scalar_t _grad_w = cache_grad_points[0], _grad_h = cache_grad_points[1], _grad_a = cache_grad_query[0];
        int sid = 2;
        for (unsigned int tid = 1; tid < blockDim.x; ++tid) {
          _grad_a += cache_grad_query[tid];
          _grad_w += cache_grad_points[sid];
          _grad_h += cache_grad_points[sid + 1];
          sid += 2;
        }

        *(grad_query + offset_query) = _grad_a;
        grad_point_w += _grad_w;
        grad_point_h += _grad_h;
      }
      __syncthreads();
    }
    *(grad_point + point_ptr) = grad_point_w;
    *(grad_point + point_ptr + 1) = grad_point_h;
  }
}


template <typename scalar_t>
__global__ void cross_ca_weight_backward_kernel_v2(const int n,
                                                   const scalar_t *grad_w,
                                                   const scalar_t *query,
                                                   const scalar_t *key,
                                                   const scalar_t *point,
                                                   scalar_t *grad_query,
                                                   scalar_t *grad_key,
                                                   scalar_t *grad_point,
                                                   const int batch_size,
                                                   const int num_heads,
                                                   const int channels,
                                                   const int height,
                                                   const int width,
                                                   const int num_query) {
  const int len_weight = (height + width - 1);
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    extern __shared__ int _s[];
    scalar_t* cache_grad_points = (scalar_t*)_s;
    scalar_t* cache_grad_query = cache_grad_points + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    const int weight_col = index % len_weight;
    const int head_col = (index / len_weight) % num_heads;
    const int query_col = (index / len_weight / num_heads) % num_query;
    const int batch_col = (index / len_weight / num_heads / num_query) % batch_size;

    const scalar_t top_grad = grad_w[index];
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    int key_base_ptr = batch_col * num_heads * channels * sp_offset + head_col * channels * sp_offset;
    int query_base_ptr = batch_col * num_query * num_heads * channels + query_col * num_heads * channels + head_col * channels;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);
    scalar_t x = 0;
    scalar_t y = 0;

    if (weight_col < width) {
      x = start_x + weight_col - 0.5;
      y = point_y - 0.5;
    } else {
      int h_index = weight_col - width;
      y = start_y + h_index;
      int j = y < point_y ? h_index : h_index + 1;
      x = point_x - 0.5;
      y = start_y + j - 0.5;
    }
    scalar_t grad_point_w = 0;
    scalar_t grad_point_h = 0;
    for (int plane = 0; plane < channels; ++plane) {
      const int offset_key = key_base_ptr + plane * sp_offset;
      const int offset_query = query_base_ptr + plane;
      const scalar_t *key_ptr = key + offset_key;
      scalar_t *grad_key_ptr = grad_key + offset_key;
      const scalar_t query_val = query[offset_query];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_query + threadIdx.x) = 0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(key_ptr, height, width, y, x, top_grad, query_val,
                             grad_key_ptr, cache_grad_query + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }
      __syncthreads();

      for (unsigned int s=blockDim.x/2, spre=blockDim.x; s>0; s>>=1, spre>>=1) {
        if (tid < s) {
          const unsigned int xid1 = tid << 1;
          const unsigned int xid2 = (tid + s) << 1;
          cache_grad_query[tid] += cache_grad_query[tid + s];
          cache_grad_points[xid1] += cache_grad_points[xid2];
          cache_grad_points[xid1 + 1] += cache_grad_points[xid2 + 1];
          if (tid + (s << 1) < spre)
          {
            cache_grad_query[tid] += cache_grad_query[tid + (s << 1)];
            cache_grad_points[xid1] += cache_grad_points[xid2 + (s << 1)];
            cache_grad_points[xid1 + 1] += cache_grad_points[xid2 + 1 + (s << 1)];
          } 
        }
        __syncthreads();
      }

      if (tid == 0) {
        *(grad_query + offset_query) = cache_grad_query[0];
        grad_point_w += cache_grad_points[0];
        grad_point_h += cache_grad_points[1];
      }
      __syncthreads();
    }
    *(grad_point + point_ptr) = grad_point_w;
    *(grad_point + point_ptr + 1) = grad_point_h;
  }
}


template <typename scalar_t>
__global__ void cross_ca_map_forward_kernel(const int n,
                                            const scalar_t *weight,
                                            const scalar_t *value,
                                            const scalar_t *point,
                                            scalar_t *output,
                                            const int batch_size,
                                            const int num_heads,
                                            const int channels,
                                            const int height,
                                            const int width,
                                            const int num_query) {
  const int len_weight = (height + width - 1);
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int c_col = index % channels;
    const int head_col = (index / channels) % num_heads;
    const int query_col = (index / channels / num_heads) % num_query;
    const int batch_col = (index / channels / num_heads / num_query) % batch_size;

    scalar_t *output_ptr = output + index;
    int weight_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * len_weight;
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    const int data_value_ptr_init_offset = batch_col  * num_heads * channels * sp_offset + head_col * channels * sp_offset + c_col * sp_offset;
    const scalar_t *data_value_ptr = value + data_value_ptr_init_offset;
    scalar_t col = 0;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);

    for (int i = 0; i < width; ++i) {
      scalar_t x = start_x + i - 0.5;
      scalar_t y = point_y - 0.5;
      const scalar_t point_weight = weight[weight_ptr + i];
      if (y > -1 && x > -1 && y < height && x < width) {
        col += bilinear_interpolate(data_value_ptr, height, width, y, x, index) * point_weight;
      }
    }
    for (int i = 0; i < height; ++i) {
      if ((start_y + i) == point_y) continue;

      int j = (start_y + i) < point_y ? i : i - 1;
      scalar_t x = point_x - 0.5;
      scalar_t y = start_y + i - 0.5;
      const scalar_t point_weight = weight[weight_ptr + width + j];
      if (y > -1 && x > -1 && y < height && x < width) {
        col += bilinear_interpolate(data_value_ptr, height, width, y, x, index) * point_weight;
      }
    }
    *output_ptr = col;
  }
}


template <typename scalar_t, unsigned int blockSize>
__global__ void cross_ca_map_backward_kernel_v1(const int n,
                                                const scalar_t *grad_out,
                                                const scalar_t *weight,
                                                const scalar_t *value,
                                                const scalar_t *point,
                                                scalar_t* grad_weight,
                                                scalar_t* grad_value,
                                                scalar_t* grad_point,
                                                const int batch_size,
                                                const int num_heads,
                                                const int channels,
                                                const int height,
                                                const int width,
                                                const int num_query) {
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_points[blockSize * 2];
    __shared__ scalar_t cache_grad_weight[blockSize];
    unsigned int tid = threadIdx.x;
    const int c_col = index % channels;
    const int head_col = (index / channels) % num_heads;
    const int query_col = (index / channels / num_heads) % num_query;
    const int batch_col = (index / channels / num_heads / num_query) % batch_size;
    const int sampling_index = index / channels;

    const scalar_t top_grad = grad_out[index];
    int weight_ptr = sampling_index * (height + width - 1);
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    const int data_value_ptr_init_offset = batch_col  * num_heads * channels * sp_offset + head_col * channels * sp_offset + c_col * sp_offset;
    const scalar_t *data_value_ptr = value + data_value_ptr_init_offset;
    scalar_t *grad_value_ptr = grad_value + data_value_ptr_init_offset;
    scalar_t *grad_point_ptr = grad_point + point_ptr;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);

    scalar_t grad_point_w = 0;
    scalar_t grad_point_h = 0;
    for (int i = 0; i < width; ++i) {
      scalar_t x = start_x + i - 0.5;
      scalar_t y = point_y - 0.5;
      const scalar_t point_weight = weight[weight_ptr + i];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_weight+threadIdx.x)=0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(data_value_ptr, height, width, y, x, top_grad, point_weight,
                             grad_value_ptr, cache_grad_weight + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }

      __syncthreads();
      if (tid == 0) {
        scalar_t _grad_w = cache_grad_points[0], _grad_h = cache_grad_points[1], _grad_a = cache_grad_weight[0];
        int sid = 2;
        for (unsigned int tid = 1; tid < blockSize; ++tid) {
          _grad_a += cache_grad_weight[tid];
          _grad_w += cache_grad_points[sid];
          _grad_h += cache_grad_points[sid + 1];
          sid += 2;
        }

        *(grad_weight + weight_ptr + i) = _grad_a;
        grad_point_w += _grad_w;
        grad_point_h += _grad_h;
      }
      __syncthreads();
    }

    for (int i = 0; i < height; ++i) {
      if ((start_y + i) == point_y) continue;

      int j = (start_y + i) < point_y ? i : i - 1;
      scalar_t x = point_x - 0.5;
      scalar_t y = start_y + i - 0.5;
      const scalar_t point_weight = weight[weight_ptr + width + j];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_weight+threadIdx.x)=0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(data_value_ptr, height, width, y, x, top_grad, point_weight,
                             grad_value_ptr, cache_grad_weight + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }
      
      __syncthreads();
      if (tid == 0) {
        scalar_t _grad_w = cache_grad_points[0], _grad_h = cache_grad_points[1], _grad_a = cache_grad_weight[0];
        int sid = 2;
        for (unsigned int tid = 1; tid < blockSize; ++tid) {
          _grad_a += cache_grad_weight[tid];
          _grad_w += cache_grad_points[sid];
          _grad_h += cache_grad_points[sid + 1];
          sid += 2;
        }

        *(grad_weight + weight_ptr + width + j) = _grad_a;
        grad_point_w += _grad_w;
        grad_point_h += _grad_h;
      }
      __syncthreads();
    }
    *(grad_point + point_ptr) = grad_point_w;
    *(grad_point + point_ptr + 1) = grad_point_h;
  }
}


template <typename scalar_t, unsigned int blockSize>
__global__ void cross_ca_map_backward_kernel_v2(const int n,
                                                const scalar_t *grad_out,
                                                const scalar_t *weight,
                                                const scalar_t *value,
                                                const scalar_t *point,
                                                scalar_t* grad_weight,
                                                scalar_t* grad_value,
                                                scalar_t* grad_point,
                                                const int batch_size,
                                                const int num_heads,
                                                const int channels,
                                                const int height,
                                                const int width,
                                                const int num_query) {
  const int sp_offset = height * width;
  CUDA_1D_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_points[blockSize * 2];
    __shared__ scalar_t cache_grad_weight[blockSize];
    unsigned int tid = threadIdx.x;
    const int c_col = index % channels;
    const int head_col = (index / channels) % num_heads;
    const int query_col = (index / channels / num_heads) % num_query;
    const int batch_col = (index / channels / num_heads / num_query) % batch_size;
    const int sampling_index = index / channels;

    const scalar_t top_grad = grad_out[index];
    int weight_ptr = sampling_index * (height + width - 1);
    int point_ptr = (batch_col * num_query * num_heads + query_col * num_heads + head_col) * 2;
    const int data_value_ptr_init_offset = batch_col  * num_heads * channels * sp_offset + head_col * channels * sp_offset + c_col * sp_offset;
    const scalar_t *data_value_ptr = value + data_value_ptr_init_offset;
    scalar_t *grad_value_ptr = grad_value + data_value_ptr_init_offset;
    scalar_t *grad_point_ptr = grad_point + point_ptr;

    const scalar_t point_x = point[point_ptr] * width;
    const scalar_t point_y = point[point_ptr + 1] * height;
    const scalar_t start_x = point_x - floor(point_x);
    const scalar_t start_y = point_y - floor(point_y);

    scalar_t grad_point_w = 0;
    scalar_t grad_point_h = 0;
    for (int i = 0; i < width; ++i) {
      scalar_t x = start_x + i - 0.5;
      scalar_t y = point_y - 0.5;
      const scalar_t point_weight = weight[weight_ptr + i];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_weight+threadIdx.x)=0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(data_value_ptr, height, width, y, x, top_grad, point_weight,
                             grad_value_ptr, cache_grad_weight + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }

      __syncthreads();

      for (unsigned int s=blockSize/2; s > 0; s>>=1) {
        if (tid < s) {
          const unsigned int xid1 = tid << 1;
          const unsigned int xid2 = (tid + s) << 1;
          cache_grad_weight[tid] += cache_grad_weight[tid + s];
          cache_grad_points[xid1] += cache_grad_points[xid2];
          cache_grad_points[xid1 + 1] += cache_grad_points[xid2 + 1];
        }
        __syncthreads();
      }

      if (tid == 0) {
        *(grad_weight + weight_ptr + i) = cache_grad_weight[0];
        grad_point_w += cache_grad_points[0];
        grad_point_h += cache_grad_points[1];
      }
      __syncthreads();
    }

    for (int i = 0; i < height; ++i) {
      if ((start_y + i) == point_y) continue;

      int j = (start_y + i) < point_y ? i : i - 1;
      scalar_t x = point_x - 0.5;
      scalar_t y = start_y + i - 0.5;
      const scalar_t point_weight = weight[weight_ptr + width + j];
      *(cache_grad_points+(threadIdx.x << 1)) = 0;
      *(cache_grad_points+((threadIdx.x << 1) + 1)) = 0;
      *(cache_grad_weight+threadIdx.x)=0;
      if (y > -1 && x > -1 && y < height && x < width) {
        cc_bilinear_gradient(data_value_ptr, height, width, y, x, top_grad, point_weight,
                             grad_value_ptr, cache_grad_weight + threadIdx.x, cache_grad_points + (threadIdx.x << 1));
      }
      
      __syncthreads();

      for (unsigned int s=blockSize/2; s > 0; s>>=1) {
        if (tid < s) {
          const unsigned int xid1 = tid << 1;
          const unsigned int xid2 = (tid + s) << 1;
          cache_grad_weight[tid] += cache_grad_weight[tid + s];
          cache_grad_points[xid1] += cache_grad_points[xid2];
          cache_grad_points[xid1 + 1] += cache_grad_points[xid2 + 1];
        }
        __syncthreads();
      }

      if (tid == 0) {
        *(grad_weight + weight_ptr + width + j) = cache_grad_weight[0];
        grad_point_w += cache_grad_points[0];
        grad_point_h += cache_grad_points[1];
      }
      __syncthreads();
    }
    *(grad_point + point_ptr) = grad_point_w;
    *(grad_point + point_ptr + 1) = grad_point_h;
  }
}


namespace detr {

at::Tensor cross_ca_weight_forward_cuda(const at::Tensor& query,
                                        const at::Tensor& key,
                                        const at::Tensor& point) {
    AT_ASSERTM(query.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(key.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(point.device().is_cuda(), "input must be a CUDA tensor");

    const int batch_size = key.size(0);
    const int num_heads = key.size(1);
    const int channels = key.size(2);
    const int height = key.size(3);
    const int width = key.size(4);

    const int num_query = point.size(1);

    auto output = at::zeros({batch_size, num_query, num_heads, height + width - 1}, query.options());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int num_kernels = batch_size * num_query * num_heads * (height + width - 1);

    AT_DISPATCH_FLOATING_TYPES(query.scalar_type(), "cross_ca_weight_forward", [&] {
        cross_ca_weight_forward_kernel<scalar_t><<<GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0, stream>>>(
          num_kernels,
          query.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, channels)
          key.contiguous().data_ptr<scalar_t>(),  // (b, num_heads, channels, height, width)
          point.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, 2)
          output.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, height + width - 1)
          batch_size, num_heads, channels, height, width, num_query);
    });

    AT_CUDA_CHECK(hipGetLastError());
    return output;
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> cross_ca_weight_backward_cuda(const at::Tensor& dw,
                                                                             const at::Tensor& query,
                                                                             const at::Tensor& key,
                                                                             const at::Tensor& point) {
    AT_ASSERTM(dw.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(query.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(key.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(point.device().is_cuda(), "input must be a CUDA tensor");

    const int batch_size = key.size(0);
    const int num_heads = key.size(1);
    const int channels = key.size(2);
    const int height = key.size(3);
    const int width = key.size(4);

    const int num_query = point.size(1);

    auto grad_query = at::zeros_like(query);
    auto grad_key = at::zeros_like(key);
    auto grad_point = at::zeros_like(point);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int num_kernels = batch_size * num_query * num_heads * (height + width - 1);
    const int num_threads = height + width - 1;

    if (num_threads < 64) {
      AT_DISPATCH_FLOATING_TYPES(query.scalar_type(), "cross_ca_weight_backward", [&] {
        cross_ca_weight_backward_kernel_v1<scalar_t>
        <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, num_threads*3*sizeof(scalar_t), stream>>>(
          num_kernels,
          dw.contiguous().data_ptr<scalar_t>(),
          query.contiguous().data_ptr<scalar_t>(),
          key.contiguous().data_ptr<scalar_t>(),
          point.contiguous().data_ptr<scalar_t>(),
          grad_query.contiguous().data_ptr<scalar_t>(),
          grad_key.contiguous().data_ptr<scalar_t>(),
          grad_point.contiguous().data_ptr<scalar_t>(),
          batch_size, num_heads, channels, height, width, num_query);
      });
    } else {
      AT_DISPATCH_FLOATING_TYPES(query.scalar_type(), "cross_ca_weight_backward", [&] {
        cross_ca_weight_backward_kernel_v2<scalar_t>
        <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, num_threads*3*sizeof(scalar_t), stream>>>(
          num_kernels,
          dw.contiguous().data_ptr<scalar_t>(),
          query.contiguous().data_ptr<scalar_t>(),
          key.contiguous().data_ptr<scalar_t>(),
          point.contiguous().data_ptr<scalar_t>(),
          grad_query.contiguous().data_ptr<scalar_t>(),
          grad_key.contiguous().data_ptr<scalar_t>(),
          grad_point.contiguous().data_ptr<scalar_t>(),
          batch_size, num_heads, channels, height, width, num_query);
      });
    }

    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_query, grad_key, grad_point);
}

at::Tensor cross_ca_map_forward_cuda(const at::Tensor& weight,
                                     const at::Tensor& value,
                                     const at::Tensor& point) {
    AT_ASSERTM(weight.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(value.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(point.device().is_cuda(), "input must be a CUDA tensor");

    const int batch_size = value.size(0);
    const int num_heads = value.size(1);
    const int channels = value.size(2);
    const int height = value.size(3);
    const int width = value.size(4);

    const int num_query = point.size(1);

    auto output = at::zeros({batch_size, num_query, num_heads, channels}, value.options());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int num_kernels = batch_size * num_query * num_heads * channels;

    AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_forward", [&] {
        cross_ca_map_forward_kernel<scalar_t><<<GET_BLOCKS(num_kernels), THREADS_PER_BLOCK, 0, stream>>>(
          num_kernels,
          weight.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, height + width - 1)
          value.contiguous().data_ptr<scalar_t>(),  // (b, num_heads, channels, height, width)
          point.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, 2)
          output.contiguous().data_ptr<scalar_t>(),  // (b, num_query, num_heads, channels)
          batch_size, num_heads, channels, height, width, num_query);
    });

    output = output.view({batch_size, num_query, num_heads*channels});
    AT_CUDA_CHECK(hipGetLastError());

    return output;
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> cross_ca_map_backward_cuda(const at::Tensor& dout,
                                                                          const at::Tensor& weight,
                                                                          const at::Tensor& value,
                                                                          const at::Tensor& point) {
    AT_ASSERTM(dout.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(value.device().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(point.device().is_cuda(), "input must be a CUDA tensor");

    const int batch_size = value.size(0);
    const int num_heads = value.size(1);
    const int channels = value.size(2);
    const int height = value.size(3);
    const int width = value.size(4);

    const int num_query = point.size(1);

    auto grad_value = at::zeros_like(value);
    auto grad_weight = at::zeros_like(weight);
    auto grad_point = at::zeros_like(point);
    auto grad_out = dout.view({batch_size, num_query, num_heads, channels});
    
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int num_kernels = batch_size * num_query * num_heads * channels;
    const int num_threads = channels;

    switch(channels) {
      case 1:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 1>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 2:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 2>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 4:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 4>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 8:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 8>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 16:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 16>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 32:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v1<scalar_t, 32>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 64:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v2<scalar_t, 64>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 128:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v2<scalar_t, 128>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 256:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v2<scalar_t, 256>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
      case 512:
        AT_DISPATCH_FLOATING_TYPES(value.scalar_type(), "cross_ca_map_backward", [&] {
          cross_ca_map_backward_kernel_v2<scalar_t, 512>
          <<<GET_BLOCKS_CUSTOM(num_kernels, num_threads), num_threads, 0, stream>>>(
            num_kernels,
            grad_out.contiguous().data_ptr<scalar_t>(),
            weight.contiguous().data_ptr<scalar_t>(),
            value.contiguous().data_ptr<scalar_t>(),
            point.contiguous().data_ptr<scalar_t>(),
            grad_weight.contiguous().data_ptr<scalar_t>(),
            grad_value.contiguous().data_ptr<scalar_t>(),
            grad_point.contiguous().data_ptr<scalar_t>(),
            batch_size, num_heads, channels, height, width, num_query);
        });
        break;
    }
    AT_CUDA_CHECK(hipGetLastError());
    return std::make_tuple(grad_value, grad_weight, grad_point);
}

}  // namespace detr
